#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <time.h>
#include<errno.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <getopt.h>
#include <hdf5.h>

#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

// Struct for header information
struct header {
  int64_t headersize,buffersize;
  unsigned int nchan,nsamp,nbit,nif,nsub;
  int machine_id,telescope_id,nbeam,ibeam,sumif;
  double tstart,tsamp,fch1,foff,fcen,bwchan;
  double src_raj,src_dej,az_start,za_start;
  char source_name[80],ifstream[8],inpfile[80];
  char *rawfname[4];
};

struct header read_h5_header(char *fname);
void get_channel_chirp(double fcen,double bw,float dm,int nchan,int nbin,int nsub,hipfftComplex *c);
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s);
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b);
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale);
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2);
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny);
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c);
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2);
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd);
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
void write_filterbank_header(struct header h,FILE *file);

// Usage
void usage()
{
  printf("cdmt -P <part> -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -o <outputname> <file.h5>\n\n");
  printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in HDF5 format.\n");
  printf("-P <part>        Specify part number for input file [integer, default: 0]\n");
  printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
  printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
  printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
  printf("-o <outputname>           Output filename [default: cdmt]\n");
  printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
  printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");

  return;
}

int main(int argc,char *argv[])
{
  int i,nsamp,nfft,mbin,nvalid,nchan=8,nbin=65536,noverlap=2048,nsub=20,ndm,ndec=1;
  int idm,iblock,nread,mchan,msamp,mblock,msum=1024;
  char *header,*h5buf[4],*dh5buf[4];
  FILE *rawfile[4],*file;
  unsigned char *cbuf,*dcbuf;
  float *fbuf,*dfbuf;
  float *bs1,*bs2,*zavg,*zstd;
  hipfftComplex *cp1,*cp2,*dc,*cp1p,*cp2p;
  hipfftHandle ftc2cf,ftc2cb;
  int idist,odist,iembed,oembed,istride,ostride;
  dim3 blocksize,gridsize;
  struct header h5;
  clock_t startclock;
  float *dm,*ddm,dm_start,dm_step;
  char fname[128],fheader[1024],*h5fname,obsid[128]="cdmt";
  int bytes_read;
  int part=0,device=0;
  int arg=0;
  FILE **outfile;

  // Read options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"P:d:D:ho:b:N:n:"))!=-1) {
      switch (arg) {
	
      case 'n':
	noverlap=atoi(optarg);
	break;

      case 'N':
	nbin=atoi(optarg);
	break;

      case 'b':
	ndec=atoi(optarg);
	break;

      case 'o':
	strcpy(obsid,optarg);
	break;
	
      case 'P':
	part=atoi(optarg);
	break;

      case 'D':
	device=atoi(optarg);
	break;
	
      case 'd':
	sscanf(optarg,"%f,%f,%d",&dm_start,&dm_step,&ndm);
	break;

      case 'h':
	usage();
	return 0;
      }
    }
  } else {
    printf("Unknown option '%c'\n", arg);
    usage();
    return 0;
  }
  h5fname=argv[optind];
  
  // Read HDF5 header
  h5=read_h5_header(h5fname);


  // Set number of subbands
  nsub=h5.nsub;

  // Adjust header for filterbank format
  h5.tsamp*=nchan*ndec;
  h5.nchan=nsub*nchan;
  h5.nbit=8;
  h5.fch1=h5.fcen+0.5*h5.nsub*h5.bwchan-0.5*h5.bwchan/nchan;
  h5.foff=-fabs(h5.bwchan/nchan);

  // Data size
  nvalid=nbin-2*noverlap;
  nsamp=100*nvalid;
  nfft=(int) ceil(nsamp/(float) nvalid);
  mbin=nbin/nchan;
  mchan=nsub*nchan;
  msamp=nsamp/nchan;
  mblock=msamp/msum;

  printf("nbin: %d nfft: %d nsub: %d mbin: %d nchan: %d nsamp: %d nvalid: %d\n",nbin,nfft,nsub,mbin,nchan,nsamp,nvalid);

  // Set device
  checkCudaErrors(hipSetDevice(device));

  // DMcK: cuFFT docs say it's best practice to plan before allocating memory
  // cuda-memcheck fails initialisation before this block is run?
  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub));
  hipDeviceSynchronize();

  // Generate FFT plan (batch in-place backward FFT)
  idist=mbin;  odist=mbin;  iembed=mbin;  oembed=mbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub));
  hipDeviceSynchronize();

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp1p,(size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2p,(size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));

  // Allocate device memory for chirp
  checkCudaErrors(hipMalloc((void **) &dc, (size_t) sizeof(hipfftComplex)*nbin*nsub*ndm));

  // Allocate device memory for block sums
  checkCudaErrors(hipMalloc((void **) &bs1, (size_t) sizeof(float)*mblock*mchan));
  checkCudaErrors(hipMalloc((void **) &bs2, (size_t) sizeof(float)*mblock*mchan));

  // Allocate device memory for channel averages and standard deviations
  checkCudaErrors(hipMalloc((void **) &zavg, (size_t) sizeof(float)*mchan));
  checkCudaErrors(hipMalloc((void **) &zstd, (size_t) sizeof(float)*mchan));

  // Allocate memory for redigitized output and header
  header=(char *) malloc(sizeof(char)*HEADERSIZE);
  for (i=0;i<4;i++) {
    h5buf[i]=(char *) malloc(sizeof(char)*nsamp*nsub);
    checkCudaErrors(hipMalloc((void **) &dh5buf[i], (size_t) sizeof(char)*nsamp*nsub));
  }

  // Allocate output buffers
  fbuf=(float *) malloc(sizeof(float)*nsamp*nsub);
  checkCudaErrors(hipMalloc((void **) &dfbuf, (size_t) sizeof(float)*nsamp*nsub));
  cbuf=(unsigned char *) malloc(sizeof(unsigned char)*msamp*mchan/ndec);
  checkCudaErrors(hipMalloc((void **) &dcbuf, (size_t) sizeof(unsigned char)*msamp*mchan/ndec));

  // Allocate DMs and copy to device
  dm=(float *) malloc(sizeof(float)*ndm);
  for (idm=0;idm<ndm;idm++)
    dm[idm]=dm_start+(float) idm*dm_step;
  checkCudaErrors(hipMalloc((void **) &ddm, (size_t) sizeof(float)*ndm));
  checkCudaErrors(hipMemcpy(ddm,dm,sizeof(float)*ndm,hipMemcpyHostToDevice));

  // Compute chirp
  blocksize.x=32; blocksize.y=32; blocksize.z=1;
  gridsize.x=nsub/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=ndm/blocksize.z+1;
  compute_chirp<<<gridsize,blocksize>>>(h5.fcen,nsub*h5.bwchan,ddm,nchan,nbin,nsub,ndm,dc);

  // Write temporary filterbank header
  file=fopen("/tmp/header.fil","w");
  write_filterbank_header(h5,file);
  fclose(file);
  file=fopen("/tmp/header.fil","r");
  bytes_read=fread(fheader,sizeof(char),1024,file);
  fclose(file);
  
  // Format file names and open
  outfile=(FILE **) malloc(sizeof(FILE *)*ndm);
  for (idm=0;idm<ndm;idm++) {
    sprintf(fname,"%s_cDM%06.2f_P%03d.fil",obsid,dm[idm],part);

    outfile[idm]=fopen(fname,"w");
  }
  
  // Write headers
  for (idm=0;idm<ndm;idm++) {
    // Send header
    fwrite(fheader,sizeof(char),bytes_read,outfile[idm]);
  }

  // Read files
  for (i=0;i<4;i++) {
    rawfile[i]=fopen(h5.rawfname[i],"r");
  }

  // Loop over input file contents
  for (iblock=0;;iblock++) {
    // Read block
    startclock=clock();
    for (i=0;i<4;i++)
      nread=fread(h5buf[i],sizeof(char),nsamp*nsub,rawfile[i])/nsub;
    if (nread==0) {
      printf("No data read from last file; assuming EOF, finishng up.\n");
      break;
    }
    printf("Block: %d: Read %d MB in %.2f s\n",iblock,sizeof(char)*nread*nsub*4/(1<<20),(float) (clock()-startclock)/CLOCKS_PER_SEC);

    // Copy buffers to device
    startclock=clock();
    for (i=0;i<4;i++)
      checkCudaErrors(hipMemcpy(dh5buf[i],h5buf[i],sizeof(char)*nread*nsub,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
    unpack_and_padd<<<gridsize,blocksize>>>(dh5buf[0],dh5buf[1],dh5buf[2],dh5buf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);

    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1p,(hipfftComplex *) cp1p,HIPFFT_FORWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2p,(hipfftComplex *) cp2p,HIPFFT_FORWARD));

    // Swap spectrum halves for large FFTs
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft*nsub/blocksize.y+1; gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1p,cp2p,nbin,nfft*nsub);

    // Loop over dms
    for (idm=0;idm<ndm;idm++) {

      // Perform complex multiplication of FFT'ed data with chirp
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=nbin*nsub/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=1;
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1p,dc,cp1,nbin*nsub,nfft,idm,1.0/(float) nbin);
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2p,dc,cp2,nbin*nsub,nfft,idm,1.0/(float) nbin);
      
      // Swap spectrum halves for small FFTs
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan*nfft*nsub/blocksize.y+1; gridsize.z=1;
      swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan*nfft*nsub);
      
      // Perform FFTs
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
      
      // Detect data
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=nfft/blocksize.z+1;
      transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan,nfft,nsub,noverlap/nchan,nread/nchan,dfbuf);
      
      // Compute block sums for redigitization
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      compute_block_sums<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,bs1,bs2);
      
      // Compute channel stats
      blocksize.x=32; blocksize.y=1; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=1; gridsize.z=1;
      compute_channel_statistics<<<gridsize,blocksize>>>(mchan,mblock,msum,bs1,bs2,zavg,zstd);

      // Redigitize data to 8bits
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      if (ndec==1)
	redigitize<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);
      else
	decimate_and_redigitize<<<gridsize,blocksize>>>(dfbuf,ndec,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);      

      // Copy buffer to host
      checkCudaErrors(hipMemcpy(cbuf,dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost));

      // Write buffer
      fwrite(cbuf,sizeof(char),nread*nsub/ndec,outfile[idm]);
    }
    printf("Processed %d DMs in %.2f s\n",ndm,(float) (clock()-startclock)/CLOCKS_PER_SEC);
  }

  // Close files
  for (i=0;i<ndm;i++)
    fclose(outfile[i]);

  // Close files
  for (i=0;i<4;i++)
    fclose(rawfile[i]);

  // Free
  free(header);
  for (i=0;i<4;i++) {
    free(h5buf[i]);
    hipFree(dh5buf);
    free(h5.rawfname[i]);
  }
  free(fbuf);
  free(dm);
  free(cbuf);
  free(outfile);

  hipFree(dfbuf);
  hipFree(dcbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(cp1p);
  hipFree(cp2p);
  hipFree(dc);
  hipFree(bs1);
  hipFree(bs2);
  hipFree(zavg);
  hipFree(zstd);
  hipFree(ddm);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}

// This is a simple H5 reader for complex voltage data. Very little
// error checking is done.
struct header read_h5_header(char *fname)
{
  int i,len,ibeam,isap;
  struct header h;
  hid_t file_id,attr_id,sap_id,beam_id,memtype,group_id,space,coord_id;
  char *string,*pch;
  const char *stokes[]={"_S0_","_S1_","_S2_","_S3_"};
  char *froot,*fpart,*ftest,group[32];
  FILE *file;

  // Find filenames
  for (i=0;i<4;i++) {
    pch=strstr(fname,stokes[i]);
    if (pch!=NULL)
      break;
  }
  len=strlen(fname)-strlen(pch);
  froot=(char *) malloc(sizeof(char)*(len+1));
  fpart=(char *) malloc(sizeof(char)*(strlen(pch)-6));
  ftest=(char *) malloc(sizeof(char)*(len+20));
  strncpy(froot,fname,len);
  strncpy(fpart,pch+4,strlen(pch)-7);

  // Check files
  for (i=0;i<4;i++) {
    // Format file name
    sprintf(ftest,"%s_S%d_%s.raw",froot,i,fpart);
    // Try to open
    if ((file=fopen(ftest,"r"))!=NULL) {
      fclose(file);
    } else {
      fprintf(stderr,"Raw file %s not found\n",ftest);
      exit (-1);
    }
    h.rawfname[i]=(char *) malloc(sizeof(char)*(strlen(ftest)+1));
    strcpy(h.rawfname[i],ftest);
  }

  // Get beam number
  for (i=0;i<4;i++) {
    pch=strstr(fname,"_B");
    if (pch!=NULL)
      break;
  }
  sscanf(pch+2,"%d",&ibeam);

  // Get SAP number
  for (i=0;i<4;i++) {
    pch=strstr(fname,"_SAP");
    if (pch!=NULL)
      break;
  }
  sscanf(pch+4,"%d",&isap);

  // Free
  free(froot);
  free(fpart);
  free(ftest);

  // Open file
  file_id=H5Fopen(fname,H5F_ACC_RDONLY,H5P_DEFAULT);

  // Open subarray pointing group
  sprintf(group,"SUB_ARRAY_POINTING_%03d",isap);
  sap_id=H5Gopen(file_id,group,H5P_DEFAULT);

  // Start MJD
  attr_id=H5Aopen(sap_id,"EXPTIME_START_MJD",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.tstart);
  H5Aclose(attr_id);

  // Declination
  attr_id=H5Aopen(sap_id,"POINT_DEC",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.src_dej);
  H5Aclose(attr_id);

  // Right ascension
  attr_id=H5Aopen(sap_id,"POINT_RA",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.src_raj);
  H5Aclose(attr_id);

  // Open beam
  sprintf(group,"BEAM_%03d",ibeam);
  beam_id=H5Gopen(sap_id,group,H5P_DEFAULT);

  // Number of samples
  attr_id=H5Aopen(beam_id,"NOF_SAMPLES",H5P_DEFAULT);
  H5Aread(attr_id,H5T_STD_U32LE,&h.nsamp);
  H5Aclose(attr_id);

  // Center frequency
  attr_id=H5Aopen(beam_id,"BEAM_FREQUENCY_CENTER",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.fcen);
  H5Aclose(attr_id);

  // Center frequency unit
  attr_id=H5Aopen(beam_id,"BEAM_FREQUENCY_CENTER_UNIT",H5P_DEFAULT);
  memtype=H5Tcopy(H5T_C_S1);
  H5Tset_size(memtype,H5T_VARIABLE);
  H5Aread(attr_id,memtype,&string);
  H5Aclose(attr_id);
  if (strcmp(string,"Hz")==0)
    h.fcen/=1e6;

  // Channel bandwidth
  attr_id=H5Aopen(beam_id,"CHANNEL_WIDTH",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.bwchan);
  H5Aclose(attr_id);

  // Center frequency unit
  attr_id=H5Aopen(beam_id,"CHANNEL_WIDTH_UNIT",H5P_DEFAULT);
  memtype=H5Tcopy(H5T_C_S1);
  H5Tset_size(memtype,H5T_VARIABLE);
  H5Aread(attr_id,memtype,&string);
  H5Aclose(attr_id);
  if (strcmp(string,"Hz")==0)
    h.bwchan/=1e6;

  // Get source
  attr_id=H5Aopen(beam_id,"TARGETS",H5P_DEFAULT);
  memtype=H5Tcopy(H5T_C_S1);
  H5Tset_size(memtype,H5T_VARIABLE);
  H5Aread(attr_id,memtype,&string);
  H5Aclose(attr_id);
  strcpy(h.source_name,string);

  // Open coordinates
  coord_id=H5Gopen(beam_id,"COORDINATES",H5P_DEFAULT);

  // Open coordinate 0
  group_id=H5Gopen(coord_id,"COORDINATE_0",H5P_DEFAULT);

  // Sampling time
  attr_id=H5Aopen(group_id,"INCREMENT",H5P_DEFAULT);
  H5Aread(attr_id,H5T_IEEE_F64LE,&h.tsamp);
  H5Aclose(attr_id);

  // Close group
  H5Gclose(group_id);

  // Open coordinate 1
  group_id=H5Gopen(coord_id,"COORDINATE_1",H5P_DEFAULT);

  // Number of subbands
  attr_id=H5Aopen(group_id,"AXIS_VALUES_WORLD",H5P_DEFAULT);
  space=H5Aget_space(attr_id);
  h.nsub=H5Sget_simple_extent_npoints(space);
  H5Aclose(attr_id);

  // Close group
  H5Gclose(group_id);

  // Close coordinates
  H5Gclose(coord_id);

  // Close beam, sap and file
  H5Gclose(beam_id);
  H5Gclose(sap_id);
  H5Fclose(file_id);

  return h;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i+nx*l]),scale);
  }
}

// Compute chirp
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c)
{
  int ibin,ichan,isub,idm,mbin,idx;
  double s,rt,t,f,fsub,fchan,bwchan,bwsub;

  // Number of channels per subband
  mbin=nbin/nchan;

  // Subband bandwidth
  bwsub=bw/nsub;

  // Channel bandwidth
  bwchan=bw/(nchan*nsub);

  // Indices of input data
  isub=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  idm=blockIdx.z*blockDim.z+threadIdx.z;

  // Keep in range
  if (isub<nsub && ichan<nchan && idm<ndm) {
    // Main constant
    s=2.0*M_PI*dm[idm]/DMCONSTANT;

    // Frequencies
    fsub=fcen-0.5*bw+bw*(float) isub/(float) nsub+0.5*bw/(float) nsub;
    fchan=fsub-0.5*bwsub+bwsub*(float) ichan/(float) nchan+0.5*bwsub/(float) nchan;
      
    // Loop over bins in channel
    for (ibin=0;ibin<mbin;ibin++) {
      // Bin frequency
      f=-0.5*bwchan+bwchan*(float) ibin/(float) mbin+0.5*bwchan/(float) mbin;
      
      // Phase delay
      rt=-f*f*s/((fchan+f)*fchan*fchan);
      
      // Taper
      t=1.0/sqrt(1.0+pow((f/(0.47*bwchan)),80));
      
      // Index
      idx=ibin+ichan*mbin+isub*mbin*nchan+idm*nsub*mbin*nchan;
      
      // Chirp
      c[idx].x=cos(rt)*t;
      c[idx].y=sin(rt)*t;
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    idx1=ibin+nbin*isub+nsub*nbin*ifft;
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    idx2=isub+nsub*isamp;
    if (isamp<0 || isamp>=nsamp) {
      cp1[idx1].x=0.0;
      cp1[idx1].y=0.0;
      cp2[idx1].x=0.0;
      cp2[idx1].y=0.0;
    } else {
      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf)
{
  int64_t ibin,ichan,ifft,isub,isamp,idx1,idx2;
  
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  ifft=blockIdx.z*blockDim.z+threadIdx.z;
  if (ibin<nbin && ichan<nchan && ifft<nfft) {
    // Loop over subbands
    for (isub=0;isub<nsub;isub++) {
      // Padded array index
      //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
      idx1=ibin+ichan*nbin+(nsub-isub-1)*nbin*nchan+ifft*nbin*nchan*nsub;

      // Time index
      isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
      
      // Output array index
      idx2=(nchan-ichan-1)+isub*nchan+nsub*nchan*isamp;
      
      // Select data points from valid region
      if (ibin>=noverlap && ibin<=nbin-noverlap && isamp>=0 && isamp<nsamp)
	fbuf[idx2]=cp1[idx1].x*cp1[idx1].x+cp1[idx1].y*cp1[idx1].y+cp2[idx1].x*cp2[idx1].x+cp2[idx1].y*cp2[idx1].y;
    }
  }

  return;
}

void send_string(const char *string,FILE *file)
{
  int len;

  len=strlen(string);
  fwrite(&len,sizeof(int),1,file);
  fwrite(string,sizeof(char),len,file);

  return;
}

void send_float(const char *string,float x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(float),1,file);

  return;
}

void send_int(const char *string,int x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(int),1,file);

  return;
}

void send_double(const char *string,double x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(double),1,file);

  return;
}

double dec2sex(double x)
{
  double d,sec,min,deg;
  char sign;
  char tmp[32];

  sign=(x<0 ? '-' : ' ');
  x=3600.0*fabs(x);

  sec=fmod(x,60.0);
  x=(x-sec)/60.0;
  min=fmod(x,60.0);
  x=(x-min)/60.0;
  deg=x;

  sprintf(tmp,"%c%02d%02d%09.6lf",sign,(int) deg,(int) min,sec);
  sscanf(tmp,"%lf",&d);

  return d;
}

void write_filterbank_header(struct header h,FILE *file)
{
  double ra,de;


  ra=dec2sex(h.src_raj/15.0);
  de=dec2sex(h.src_dej);
  
  send_string("HEADER_START",file);
  send_string("rawdatafile",file);
  send_string(h.rawfname[0],file);
  send_string("source_name",file);
  send_string(h.source_name,file);
  send_int("machine_id",11,file);
  send_int("telescope_id",11,file);
  send_double("src_raj",ra,file);
  send_double("src_dej",de,file);
  send_int("data_type",1,file);
  send_double("fch1",h.fch1,file);
  send_double("foff",h.foff,file);
  send_int("nchans",h.nchan,file);
  send_int("nbeams",0,file);
  send_int("ibeam",0,file);
  send_int("nbits",h.nbit,file);
  send_double("tstart",h.tstart,file);
  send_double("tsamp",h.tsamp,file);
  send_int("nifs",1,file);
  send_string("HEADER_END",file);

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2)
{
  int64_t ichan,iblock,isum,idx1,idx2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    idx1=ichan+nchan*iblock;
    bs1[idx1]=0.0;
    bs2[idx1]=0.0;
    for (isum=0;isum<nsum;isum++) {
      idx2=ichan+nchan*(isum+iblock*nsum);
      bs1[idx1]+=z[idx2];
      bs2[idx1]+=z[idx2]*z[idx2];
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd)
{
  int64_t ichan,iblock,idx1;
  double s1,s2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  if (ichan<nchan) {
    s1=0.0;
    s2=0.0;
    for (iblock=0;iblock<nblock;iblock++) {
      idx1=ichan+nchan*iblock;
      s1+=bs1[idx1];
      s2+=bs2[idx1];
    }
    zavg[ichan]=s1/(float) (nblock*nsum);
    zstd[ichan]=s2/(float) (nblock*nsum)-zavg[ichan]*zavg[ichan];
    zstd[ichan]=sqrt(zstd[ichan]);
  }

  return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1;
  float zoffset,zscale;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum++) {
      idx1=ichan+nchan*(isum+iblock*nsum);
      z[idx1]-=zoffset;
      z[idx1]*=256.0/zscale;
      cz[idx1]=(unsigned char) z[idx1];
      if (z[idx1]<0.0) cz[idx1]=0;
      if (z[idx1]>255.0) cz[idx1]=255;
    }
  }

  return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float zoffset,zscale,ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
	idx1=ichan+nchan*(isum+idec+iblock*nsum);
	ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      ztmp-=zoffset;
      ztmp*=256.0/zscale;
      cz[idx2]=(unsigned char) ztmp;
      if (ztmp<0.0) cz[idx2]=0;
      if (ztmp>255.0) cz[idx2]=255;
    }
  }

  return;
}
