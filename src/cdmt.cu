#include "hip/hip_runtime.h"
#include "cdmt.h"

int main(int argc,char *argv[])
{
  int arg=0;             // Command line inputs
  int i,idm,iblock;      // Iterators
  clock_t startclock;    // Clock time
  size_t cfsize, cbsize; // FFT sizes
  size_t minfftsize;     // Min of cfsize and cbsize
  size_t gpu_mems[2];    // Used/total GPU VRAM

  // File input
  char *h5fname;        // Name of first HDF5 metadata file
  FILE *file;           // Pointer to first HDF5 metadata file
  FILE *input_files[4]; // Pointers to all HDF5 raw file parts
  struct header h5;     // Struct containing HDF5 header info
  char fheader[1024];   // Filterbank header string
  int nread,nread_tmp;  // Amount of data read from block in bytes
  int bytes_read;       // Filterbank header size in bytes
  char *h5buf[4];       // Host buffer for reading HDF5 data
  char *dh5buf[4];      // Device buffer for reading HDF5 data
  
  // File output
  char fname[256];      // Name of output filterbanks
  char obsid[128]="cdmt";  // Prefix of the output filenames
  FILE **output_files;       // Pointer to filterbank output_filess
  float *dfbuf;         // Device buffer for output
  unsigned char *dcbuf; // Device buffer for redigitised output
  unsigned char *cbuf;  // Host buffer for redigitised output

  // DMs
  float *dm,*ddm;      // Host/device arrays for DM steps
  float dm_start=-1;   // Start DM
  float dm_step=-1;    // DM step size
  int ndm=-1;          // Number of DM steps

  // Forward FFT
  int nforward=128;   // Number of forward FFTs per cuFFT call
  int nsub=20;        // Number of subbands
  int nchan=8;        // Number of channels per subband
  int nbin=65536;     // Size of forward FFT
  int noverlap=2048;  // Size of the overlap region
  int nvalid;         // Number of non-overlapping bins
  int nsamp;          // Number of samples per block
  int nfft;           // Number of parallel FFTs
  int ndec=1;         // Number of time samples to average

  // Backward FFT
  int mchan;          // Number of filterbank channels (nsub*nchan)
  int mbin;           // Size of backward FFT (nbin/nchan)
  int msamp;          // Number of block samples per channel (nsamp/nchan)
  int msum=1024;      // Size of block sum
  int mblock;         // Number of blocks (msamp/msum)

  // CUDA
  int device=0;             // GPU device code
  hipfftComplex *cp1p,*cp2p; // Complex timeseries
  hipfftComplex *cp1,*cp2;   // Dedispersed complex timeseries
  hipfftComplex *dc;         // Chirp
  float *bs1,*bs2;          // Block sums
  float *zavg,*zstd;        // Channel averages and standard deviations
  hipfftHandle ftc2cf;       // Forward FFT plan
  hipfftHandle ftc2cb;       // Backward FFT plan
  int idist,odist,iembed,oembed,istride,ostride;  // FFT plan params
  dim3 blocksize,gridsize;  // GPU mapping params

  // Read options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"hd:D:b:N:n:f:s:c:m:o:"))!=-1) {
      switch (arg) {

        case 'h':
          usage();
          return 0;

        case 'd':
          sscanf(optarg,"%f,%f,%d",&dm_start,&dm_step,&ndm);
          break;

        case 'D':
          device=atoi(optarg);
          break;
        
        case 'b':
          ndec=atoi(optarg);
          break;

        case 'N':
          nbin=atoi(optarg);
          break;
	
        case 'n':
          noverlap=atoi(optarg);
          break;

        case 'f':
          nforward=atoi(optarg);
          break;

        case 's':
          nsub=atoi(optarg);
          break;

        case 'c':
          nchan=atoi(optarg);
          break;

        case 'm':
          msum=atoi(optarg);
          break;

        case 'o':
          strcpy(obsid,optarg);
          break;

        default:
          return 1;
      }
    }
  } else {
    usage();
    return 0;
  }

  // Check required inputs were given
  if ((dm_start==-1)||(dm_step==-1)||(ndm==-1)) {
    fprintf(stderr, "ERROR :: DM parameters were not specified. Exiting.\n");
    return 1;
  }
  if (argc<=optind) {
    fprintf(stderr, "ERROR :: Failed to provide an input file. Exiting.\n");
    return 1;
  }
  h5fname=argv[optind];

  // Basic input checks
  if (dm_start<0.0) {
    fprintf(stderr, "ERROR :: Start DM must be a non-negative number (currently %f). Exiting.\n", dm_start);
    return 1;
  }
  if (dm_step<=0.0) {
    fprintf(stderr, "ERROR :: DM step size must be a positive number (currently %f). Exiting.\n", dm_step);
    return 1;
  }
  if (ndm<1) {
    fprintf(stderr, "ERROR :: Number of DM trials must be a positive integer (currently %d). Exiting.\n", ndm);
    return 1;
  }
  if (ndec<1) {
    fprintf(stderr, "ERROR :: Number of averages time samples must be a positive integer (currently %d). Exiting.\n", ndec);
    return 1;
  }
  if (nbin<1) {
    fprintf(stderr, "ERROR :: FFT size must be a positive integer (currently %d). Exiting.\n", nbin);
    return 1;
  }
  if (noverlap<1) {
    fprintf(stderr, "ERROR :: FFT overlap must be a positive integer (currently %d). Exiting.\n", noverlap);
    return 1;
  }
  if (nforward<1) {
    fprintf(stderr, "ERROR :: Number of FFTs must be a positive integer (currently %d). Exiting.\n", nforward);
    return 1;
  }
  if (nsub<1) {
    fprintf(stderr, "ERROR :: Number of subbands must be a positive integer (currently %d). Exiting.\n", nsub);
    return 1;
  }
  if (nchan<1) {
    fprintf(stderr, "ERROR :: Channelisation factor must be a positive integer (currently %d). Exiting.\n", nchan);
    return 1;
  }
  if (msum<1) {
    fprintf(stderr, "ERROR :: Size of blocksum must be a positive integer (currently %d). Exiting.\n", msum);
    return 1;
  }

  // Sanity checks
  if (nbin % nchan != 0) {
    fprintf(stderr, "ERROR :: nbin must be divisible by nchan (%d) (currently %d, remainder: %d). Exiting.\n", nchan, nbin, nbin % nchan);
    return 1;
  }
  if (nbin-2*noverlap < 1) {
    fprintf(stderr, "ERROR :: FFT size (%d) must be greater than twice the FFT overlap (%d). Exiting.\n", nbin, noverlap);
    return 1;
  }
  if ((nforward * (nbin-2*noverlap)) % nchan != 0) {
    fprintf(stderr, "ERROR :: Number of valid samples must be divisible by nchan (%d) (currently %d, remainer %d). Exiting.\n", nchan, nbin-2*noverlap, (nbin-2*noverlap) % nchan);
    return 1;
  }
  if ((nforward * (nbin-2*noverlap) / nchan) % msum != 0) {
    fprintf(stderr, "ERROR :: Number of valid samples must be divisible by msum (%d) (currently %d, remainder %d).\n", msum, (nforward * (nbin-2*noverlap) / nchan), (nforward * (nbin-2*noverlap) / nchan) % msum);
    return 1;
  }
  if ((nforward * (nbin-2*noverlap)) % 128 != 0) {
    fprintf(stderr, "ERROR :: Number of valid samples must be divisible by samples per packet (128) (currently %d, remainder %d). Exiting.\n", (nforward * (nbin-2*noverlap)), (nforward * (nbin-2*noverlap)) % 128);
    return 1;
  }

  // File checks
   if (access(h5fname, F_OK)==-1)
  {
    fprintf(stderr, "ERROR :: Input file does not exist (%s). Exiting.\n", h5fname);
    return 1;
  }
  if (access(h5fname, R_OK)==-1)
  {
    fprintf(stderr, "ERROR :: Input file is not readable (%s). Exiting.\n", h5fname);
    return 1;
  }

  // Read HDF5 header
  h5=read_h5_header(h5fname);

  // Check that the FFT size and overlap size are large enough
  const double  stg1 = (1.0 / 2.41e-4) * abs(pow((double) h5.fch1 + h5.nsub * h5.foff + h5.foff *0.5,-2.0) - pow((double) h5.fch1 + h5.nsub * h5.foff - h5.foff *0.5, -2.0)) * (dm_start + dm_step * (ndm - 1));
  const int overlap_check = (int) (stg1 / h5.tsamp);
  if (overlap_check > nbin) {
    fprintf(stderr, "WARNING :: The size of your FFT bin is too short for the given DMs and frequencies. Given bin size: %d, Suggested minimum bin size: %d (maximum dispersion delay %f).\n", nbin, overlap_check, stg1);
  } else if (overlap_check / 2 > noverlap) {
    fprintf(stderr, "WARNING :: The size of your FFT overlap is too short for the given maximum DM. Given overlap: %d, Suggested minimum overlap: %d (maximum dispersion delay %f).\n", noverlap, overlap_check / 2, stg1);
  }

  // Open input data files
  for (i=0;i<4;i++) {
    printf("Opening file %s\n", h5.rawfname[i]);
    input_files[i]=fopen(h5.rawfname[i],"r");
    if (input_files[i]==NULL) {
      fprintf(stderr, "ERROR :: Input file failed to open (null pointer). Exiting.\n");
      return 1;
    }
  }

  // Set number of subbands
  nsub=h5.nsub;

  // Adjust header for filterbank format
  h5.tsamp*=nchan*ndec;
  h5.nchan=nsub*nchan;
  h5.nbit=8;
  h5.fch1=h5.fcen+0.5*h5.nsub*h5.bwchan-0.5*h5.bwchan/nchan;
  h5.foff=-fabs(h5.bwchan/nchan);

  // Data sizes
  nvalid=nbin-2*noverlap;
  nsamp=nforward*nvalid;
  nfft=(int) ceil(nsamp/(float) nvalid);
  mbin=nbin/nchan;  // nbin must be divisible by nchan
  mchan=nsub*nchan;
  msamp=nsamp/nchan;  // nforward * nvalid must be divisible by nchan
  mblock=msamp/msum;  // nforward * nvalid / nchan must be divisible by msum

  printf("\nUsing the following parameters:\n\n");
  printf("         Num of subbands = %d\n", nsub);
  printf("   Channelisation factor = %d\n", nchan);
  printf("     Downsampling factor = %d\n", ndec);
  printf("        Forward FFT size = %d\n", nbin);
  printf("       Backward FFT size = %d\n", mbin);
  printf("   Valid samples per FFT = %d\n", nvalid);
  printf("             FFTs per op = %d\n", nfft);
  printf("  Samples per forward op = %d\n", nsamp);
  printf(" Samples per backward op = %d\n", msamp);
  printf("       Size of block sum = %d\n", msum);
  printf("       Num of block sums = %d\n\n", mblock);

  // Set device
  checkCudaErrors(hipSetDevice(device));

  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCufftErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub));
  checkCufftErrors(hipfftGetSizeMany(ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub,&cfsize));

  // Generate FFT plan (batch in-place backward FFT)
  idist=mbin;  odist=mbin;  iembed=mbin;  oembed=mbin;  istride=1;  ostride=1;
  checkCufftErrors(hipfftPlanMany(&ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub));
  checkCufftErrors(hipfftGetSizeMany(ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub,&cbsize));

  // Get the maximum size needed for the FFT operations (they should be the same, check for safety)
  minfftsize = cfsize > cbsize ? cfsize : cbsize;
  printf("Allocated %ld MB for cuFFT work (saving %ld MB)\n", minfftsize >> 20, (cfsize + cbsize - minfftsize) >> 20);

  // Predict the overall VRAM usage
  long unsigned int bytes_used=\
      sizeof(hipfftComplex)*nbin*nfft*nsub*4 \
    + sizeof(hipfftComplex)*nbin*nsub*ndm \
    + sizeof(float)*mblock*mchan*2 \
    + sizeof(char)*nsamp*nsub*4 \
    + sizeof(float)*nsamp*nsub \
    + sizeof(unsigned char)*msamp*mchan/ndec \
    + sizeof(float)*ndm;

  // Get the total / available VRAM
  checkCudaErrors(hipMemGetInfo(&(gpu_mems[0]), &(gpu_mems[1])));
  printf("Preparing for GPU memory allocations. Current memory usage: %ld / %ld GB\n", (gpu_mems[1] - gpu_mems[0]) >> 30, gpu_mems[1] >> 30);
  printf("We anticipate %ld MB (%ld GB) to be allocated on the GPU (%ld MB for cuFFT planning).\n", (bytes_used + minfftsize) >> 20, (bytes_used + minfftsize) >> 30, minfftsize >> 20);

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp1p,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2p,sizeof(hipfftComplex)*nbin*nfft*nsub));

  // Allocate device memory for chirp
  checkCudaErrors(hipMalloc((void **) &dc,sizeof(hipfftComplex)*nbin*nsub*ndm));

  // Allocate device memory for block sums
  checkCudaErrors(hipMalloc((void **) &bs1,sizeof(float)*mblock*mchan));
  checkCudaErrors(hipMalloc((void **) &bs2,sizeof(float)*mblock*mchan));

  // Allocate device memory for channel averages and standard deviations
  checkCudaErrors(hipMalloc((void **) &zavg,sizeof(float)*mchan));
  checkCudaErrors(hipMalloc((void **) &zstd,sizeof(float)*mchan));

  // Allocate memory for redigitized output and header
  for (i=0;i<4;i++) {
    h5buf[i]=(char *) malloc(sizeof(char)*nsamp*nsub);
    checkCudaErrors(hipMalloc((void **) &dh5buf[i],sizeof(char)*nsamp*nsub));
  }

  // Allocate output buffers for final data products
  checkCudaErrors(hipMalloc((void **) &dfbuf,sizeof(float)*nsamp*nsub));
  cbuf=(unsigned char *) malloc(sizeof(unsigned char)*msamp*mchan/ndec);
  checkCudaErrors(hipMalloc((void **) &dcbuf,sizeof(unsigned char)*msamp*mchan/ndec));

  // Allocate DMs and copy to device
  dm=(float *) malloc(sizeof(float)*ndm);
  for (idm=0;idm<ndm;idm++)
    dm[idm]=dm_start+(float) idm*dm_step;
  checkCudaErrors(hipMalloc((void **) &ddm,sizeof(float)*ndm));
  checkCudaErrors(hipMemcpy(ddm,dm,sizeof(float)*ndm,hipMemcpyHostToDevice));

  // Allow memory alloation/copy actions to finish before processing
  hipDeviceSynchronize();

  // Compute chirp
  blocksize.x=32; blocksize.y=32; blocksize.z=1;
  gridsize.x=nsub/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=ndm/blocksize.z+1;
  compute_chirp<<<gridsize,blocksize>>>(h5.fcen,nsub*h5.bwchan,ddm,nchan,nbin,nsub,ndm,dc);

  // Write temporary filterbank header
  file=fopen("/tmp/header.fil","w");
  if (file==NULL) {
    fprintf(stderr, "ERROR :: Unable to open /tmp/header.fil to write temporary header. Exiting.\n");
    return 1;
  }
  write_filterbank_header(h5,file);
  fclose(file);
  file=fopen("/tmp/header.fil","r");
  if (file==NULL) {
    fprintf(stderr, "ERROR :: Unable to open /tmp/header.fil to read temporary header length. Exiting.\n");
    return 1;
  }
  bytes_read=fread(fheader,sizeof(char),1024,file);
  fclose(file);
  
  // Format file names and open
  output_files=(FILE **) malloc(sizeof(FILE *)*ndm);
  for (idm=0;idm<ndm;idm++) {
    sprintf(fname,"%s_cDM%06.2f.fil",obsid,dm[idm]);

    output_files[idm]=fopen(fname,"w");
    if (output_files[idm]==NULL) {
      fprintf(stderr, "ERROR :: Unable to open output file %s. Exiting.\n", fname);
      return 1;
    }
  }

  // Write headers
  for (idm=0;idm<ndm;idm++) {
    // Send header
    fwrite(fheader,sizeof(char),bytes_read,output_files[idm]);
  }

  // Loop over input file contents
  nread=INT_MAX;
  for (iblock=0;;iblock++) {
    // Read block
    startclock=clock();
    for (i=0;i<4;i++)
      nread_tmp=fread(h5buf[i],sizeof(char),nsamp*nsub,input_files[i])/nsub;

    if (nread > nread_tmp) {
      nread = nread_tmp;
    }

    printf("Block: %d: Read %lu MB in %.2f s\n",iblock,sizeof(char)*nread*nsub*4/(1<<20),(float) (clock()-startclock)/CLOCKS_PER_SEC);

    if (nread==0) {
      printf("No data read from last block; assuming EOF, finishing up.\n");
      break;
    } else if (iblock != 0 && nread < nread_tmp) {
      printf("Received less data than expected; we may have parsed out of order data or we are nearing the EOF.\n");
    }      

    // Copy buffers to device
    startclock=clock();
    for (i=0;i<4;i++)
      checkCudaErrors(hipMemcpy(dh5buf[i],h5buf[i],sizeof(char)*nread*nsub,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
    if (iblock>0) {
      unpack_and_padd<<<gridsize,blocksize>>>(dh5buf[0],dh5buf[1],dh5buf[2],dh5buf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);
    } else {
      unpack_and_padd_first_iteration<<<gridsize,blocksize>>>(dh5buf[0],dh5buf[1],dh5buf[2],dh5buf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);
    }

    // Perform FFTs
    checkCufftErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1p,(hipfftComplex *) cp1p,HIPFFT_FORWARD));
    checkCufftErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2p,(hipfftComplex *) cp2p,HIPFFT_FORWARD));

    // Swap spectrum halves for large FFTs
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft*nsub/blocksize.y+1; gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1p,cp2p,nbin,nfft*nsub);

    // Loop over dms
    for (idm=0;idm<ndm;idm++) {

      // Perform complex multiplication of FFT'ed data with chirp
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=nbin*nsub/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=1;
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1p,dc,cp1,nbin*nsub,nfft,idm,1.0/(float) nbin);
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2p,dc,cp2,nbin*nsub,nfft,idm,1.0/(float) nbin);

      // Padd the next iteration
      if (idm==ndm-1) {
        blocksize.x=32; blocksize.y=32; blocksize.z=1;
        gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
        padd_next_iteration<<<gridsize,blocksize>>>(dh5buf[0],dh5buf[1],dh5buf[2],dh5buf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);
      }
      
      // Swap spectrum halves for small FFTs
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan*nfft*nsub/blocksize.y+1; gridsize.z=1;
      swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan*nfft*nsub);
      
      // Perform FFTs
      checkCufftErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
      checkCufftErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
      
      // Detect data
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=nfft/blocksize.z+1;
      transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan,nfft,nsub,noverlap/nchan,nread/nchan,dfbuf);
      
      // Compute block sums for redigitization
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      compute_block_sums<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,bs1,bs2);
      
      // Compute channel stats
      blocksize.x=32; blocksize.y=1; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=1; gridsize.z=1;
      compute_channel_statistics<<<gridsize,blocksize>>>(mchan,mblock,msum,bs1,bs2,zavg,zstd);

      // Redigitize data to 8bits
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      if (ndec==1)
	      redigitize<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);
      else
	      decimate_and_redigitize<<<gridsize,blocksize>>>(dfbuf,ndec,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);      

      // Copy buffer to host
      checkCudaErrors(hipMemcpy(cbuf,dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost));

      // Write buffer
      fwrite(cbuf,sizeof(char),nread*nsub/ndec,output_files[idm]);
    }
    printf("Processed %d DMs in %.2f s\n",ndm,(float) (clock()-startclock)/CLOCKS_PER_SEC);
  }

  // Close files
  for (i=0;i<ndm;i++)
    fclose(output_files[i]);

  // Close files
  for (i=0;i<4;i++)
    fclose(input_files[i]);

  // Free
  for (i=0;i<4;i++) {
    free(h5buf[i]);
    hipFree(dh5buf);
    free(h5.rawfname[i]);
  }
  free(dm);
  free(cbuf);
  free(output_files);

  hipFree(dfbuf);
  hipFree(dcbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(cp1p);
  hipFree(cp2p);
  hipFree(dc);
  hipFree(bs1);
  hipFree(bs2);
  hipFree(zavg);
  hipFree(zstd);
  hipFree(ddm);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}

void usage()
{
  printf("CDMT - Coherent Dispersion Measure Trials\n");
  printf("Compute coherently dedispersed SIGPROC filterbank files from complex voltage data.\n\n");
  printf("Usage:\n");
  printf("  cdmt [options...] [header_file]\n\n");
  printf("Arguments:\n");
  printf("  header_file             The header file of the lowest subband\n\n");
  printf("Options:\n");
  printf("  -d <DM start,step,num>  DM start, stepsize, and number of trials\n");
  printf("  -D <GPU device>         GPU device number to use (default: 0)\n");
  printf("  -b <ndec>               Number of time samples to average (default: 1)\n");
  printf("  -N <forward FFT size>   Forward FFT size (default: 32768)\n");
  printf("  -n <overlap size>       FFT overlap size (default: 1024)\n");
  printf("  -f <FFTs per op>        Number of FFTs per cuFFT call (default: 128)\n");
  printf("  -s <nsub>               Number of input subbands (default: 24)\n");
  printf("  -c <nchan>              Channelisation factor (default: 128)\n");
  printf("  -m <msum>               Size of blocksum (default: 1920)\n");
  printf("  -o <output prefix>      Output filename prefix (default: cdmt)\n");
  return;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i+nx*l]),scale);
  }
}

// Compute chirp
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c)
{
  int ibin,ichan,isub,idm,mbin,idx;
  double s,rt,t,f,fsub,fchan,bwchan,bwsub;

  // Number of channels per subband
  mbin=nbin/nchan;

  // Subband bandwidth
  bwsub=bw/nsub;

  // Channel bandwidth
  bwchan=bw/(nchan*nsub);

  // Indices of input data
  isub=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  idm=blockIdx.z*blockDim.z+threadIdx.z;

  // Keep in range
  if (isub<nsub && ichan<nchan && idm<ndm) {
    // Main constant
    s=2.0*M_PI*dm[idm]/DMCONSTANT;

    // Frequencies
    fsub=fcen-0.5*bw+bw*(float) isub/(float) nsub+0.5*bw/(float) nsub;
    fchan=fsub-0.5*bwsub+bwsub*(float) ichan/(float) nchan+0.5*bwsub/(float) nchan;
      
    // Loop over bins in channel
    for (ibin=0;ibin<mbin;ibin++) {
      // Bin frequency
      f=-0.5*bwchan+bwchan*(float) ibin/(float) mbin+0.5*bwchan/(float) mbin;
      
      // Phase delay
      rt=-f*f*s/((fchan+f)*fchan*fchan);
      
      // Taper
      t=1.0/sqrt(1.0+pow((f/(0.47*bwchan)),80));
      
      // Index
      idx=ibin+ichan*mbin+isub*mbin*nchan+idm*nsub*mbin*nchan;
      
      // Chirp
      c[idx].x=cos(rt)*t;
      c[idx].y=sin(rt)*t;
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    if (isamp >= noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(isamp-noverlap);

      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution. This is separate from the main kernel to minimise performance
// loss to branching on the GPU. On the first iteration, we want to fill
// the final non-noverlap region and final noverlap region so that they can 
// match the first noverlap region and first non-noverlap on the second
// iteration
__global__ void unpack_and_padd_first_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    if (isamp >= noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(isamp-noverlap);

      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    } else if (isamp > -noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(noverlap-isamp);

      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are located in the first noverlap region and first non-
// noverlap region, for continuous time series between data blocks
// 
// overlap_(timeblock)_(index)
// t = 0: overlap_0_0: nfft_0_0, nfft_0_1... nfft_0_N-1, nfft_0 N: overlap_0_1
// t = 1: nfft_0_N: overlap_0_1, nfft_1_0.... nfft_1_N-1:overlap_1_1
// t = 2 nfft_1_N-1: overlap_1_1...
// etc
__global__ void padd_next_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft;
    if (isamp<2*noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(isamp+nsamp-2*noverlap);

      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf)
{
  int64_t ibin,ichan,ifft,isub,isamp,idx1,idx2;
  
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  ifft=blockIdx.z*blockDim.z+threadIdx.z;
  if (ibin<nbin && ichan<nchan && ifft<nfft) {
    // Loop over subbands
    for (isub=0;isub<nsub;isub++) {
      // Padded array index
      //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
      idx1=ibin+ichan*nbin+(nsub-isub-1)*nbin*nchan+ifft*nbin*nchan*nsub;

      // Time index
      isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
      
      // Output array index
      idx2=(nchan-ichan-1)+isub*nchan+nsub*nchan*isamp;
      
      // Select data points from valid region
      if (ibin>=noverlap && ibin<=nbin-noverlap && isamp>=0 && isamp<nsamp)
	fbuf[idx2]=cp1[idx1].x*cp1[idx1].x+cp1[idx1].y*cp1[idx1].y+cp2[idx1].x*cp2[idx1].x+cp2[idx1].y*cp2[idx1].y;
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2)
{
  int64_t ichan,iblock,isum,idx1,idx2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    idx1=ichan+nchan*iblock;
    bs1[idx1]=0.0;
    bs2[idx1]=0.0;
    for (isum=0;isum<nsum;isum++) {
      idx2=ichan+nchan*(isum+iblock*nsum);
      bs1[idx1]+=z[idx2];
      bs2[idx1]+=z[idx2]*z[idx2];
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd)
{
  int64_t ichan,iblock,idx1;
  double s1,s2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  if (ichan<nchan) {
    s1=0.0;
    s2=0.0;
    for (iblock=0;iblock<nblock;iblock++) {
      idx1=ichan+nchan*iblock;
      s1+=bs1[idx1];
      s2+=bs2[idx1];
    }
    zavg[ichan]=s1/(float) (nblock*nsum);
    zstd[ichan]=s2/(float) (nblock*nsum)-zavg[ichan]*zavg[ichan];
    zstd[ichan]=sqrt(zstd[ichan]);
  }

  return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1;
  float zoffset,zscale;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum++) {
      idx1=ichan+nchan*(isum+iblock*nsum);
      z[idx1]-=zoffset;
      z[idx1]*=256.0/zscale;
      cz[idx1]=(unsigned char) z[idx1];
      if (z[idx1]<0.0) cz[idx1]=0;
      if (z[idx1]>255.0) cz[idx1]=255;
    }
  }

  return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float zoffset,zscale,ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
	idx1=ichan+nchan*(isum+idec+iblock*nsum);
	ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      ztmp-=zoffset;
      ztmp*=256.0/zscale;
      cz[idx2]=(unsigned char) ztmp;
      if (ztmp<0.0) cz[idx2]=0;
      if (ztmp>255.0) cz[idx2]=255;
    }
  }

  return;
}
