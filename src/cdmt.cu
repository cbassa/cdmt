#include "hip/hip_runtime.h"
#include "cdmt.h"

int main(int argc,char *argv[])
{
  int arg=0;            // Command line inputs
  int i,idm,iblock;     // Iterators
  clock_t startclock;   // Clock time

  // File input
  char *h5fname;        // Name of first HDF5 metadata file
  FILE *file;           // Pointer to first HDF5 metadata file
  FILE *rawfile[4];     // Pointers to all HDF5 raw file parts
  struct header h5;     // Struct containing HDF5 header info
  char fheader[1024];   // Filterbank header string
  int nread;            // Amount of data read from block in bytes
  int bytes_read;       // Filterbank header size in bytes
  char *h5buf[4];       // Host buffer for reading HDF5 data
  char *dh5buf[4];      // Device buffer for reading HDF5 data
  
  // File output
  char fname[462];      // Name of output filterbanks
  char obsid[128]="cdmt";  // Prefix of the output filenames
  int part=0;           // Part number (used for naming output files)
  FILE **outfile;       // Pointer to filterbank outfiles
  float *dfbuf;         // Device buffer for output
  unsigned char *dcbuf; // Device buffer for redigitised output
  unsigned char *cbuf;  // Host buffer for redigitised output

  // DMs
  float *dm,*ddm;      // Host/device arrays for DM steps
  float dm_start;      // Start DM
  float dm_step;       // DM step size
  int ndm;             // Number of DM steps

  // Forward FFT
  int nsub=20;        // Number of subbands
  int nchan=8;        // Number of channels per subband
  int nbin=65536;     // Size of forward FFT
  int noverlap=2048;  // Size of the overlap region
  int nvalid;         // Number of non-overlapping bins
  int nsamp;          // Number of samples per block
  int nfft;           // Number of parallel FFTs
  int ndec=1;         // Number of time samples to average

  // Backward FFT
  int mchan;          // Number of filterbank channels (nsub*nchan)
  int mbin;           // Size of backward FFT (nbin/nchan)
  int msamp;          // Number of block samples per channel (nsamp/nchan)
  int msum=1024;      // Size of block sum
  int mblock;         // Number of blocks (msamp/msum)

  // CUDA
  int device=0;             // GPU device code
  hipfftComplex *cp1p,*cp2p; // Complex timeseries
  hipfftComplex *cp1,*cp2;   // Dedispersed complex timeseries
  hipfftComplex *dc;         // Chirp
  float *bs1,*bs2;          // Block sums
  float *zavg,*zstd;        // Channel averages and standard deviations
  hipfftHandle ftc2cf;       // Forward FFT plan
  hipfftHandle ftc2cb;       // Backward FFT plan
  int idist,odist,iembed,oembed,istride,ostride;  // FFT plan params
  dim3 blocksize,gridsize;  // GPU mapping params

  // Read options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"P:d:D:ho:b:N:n:"))!=-1) {
      switch (arg) {
	
        case 'n':
          noverlap=atoi(optarg);
          break;

        case 'N':
          nbin=atoi(optarg);
          break;

        case 'b':
          ndec=atoi(optarg);
          break;

        case 'o':
          strcpy(obsid,optarg);
          break;
    
        case 'P':
          part=atoi(optarg);
          break;

        case 'D':
          device=atoi(optarg);
          break;
    
        case 'd':
          sscanf(optarg,"%f,%f,%d",&dm_start,&dm_step,&ndm);
          break;

        case 'h':
          usage();
          return 0;
      }
    }
  } else {
    usage();
    return 0;
  }

  // Parse the HDF5 file name
  h5fname=argv[optind];
  
  // Read HDF5 header
  h5=read_h5_header(h5fname);

  // Set number of subbands
  nsub=h5.nsub;

  // Adjust header for filterbank format
  h5.tsamp*=nchan*ndec;
  h5.nchan=nsub*nchan;
  h5.nbit=8;
  h5.fch1=h5.fcen+0.5*h5.nsub*h5.bwchan-0.5*h5.bwchan/nchan;
  h5.foff=-fabs(h5.bwchan/nchan);

  // Data size
  nvalid=nbin-2*noverlap;
  nsamp=100*nvalid;
  nfft=(int) ceil(nsamp/(float) nvalid);
  mbin=nbin/nchan;
  mchan=nsub*nchan;
  msamp=nsamp/nchan;
  mblock=msamp/msum;

  printf("nbin: %d nfft: %d nsub: %d mbin: %d nchan: %d nsamp: %d nvalid: %d\n",nbin,nfft,nsub,mbin,nchan,nsamp,nvalid);

  // Set device
  checkCudaErrors(hipSetDevice(device));

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp1p,sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2p,sizeof(hipfftComplex)*nbin*nfft*nsub));

  // Allocate device memory for chirp
  checkCudaErrors(hipMalloc((void **) &dc,sizeof(hipfftComplex)*nbin*nsub*ndm));

  // Allocate device memory for block sums
  checkCudaErrors(hipMalloc((void **) &bs1,sizeof(float)*mblock*mchan));
  checkCudaErrors(hipMalloc((void **) &bs2,sizeof(float)*mblock*mchan));

  // Allocate device memory for channel averages and standard deviations
  checkCudaErrors(hipMalloc((void **) &zavg,sizeof(float)*mchan));
  checkCudaErrors(hipMalloc((void **) &zstd,sizeof(float)*mchan));

  // Allocate memory for redigitized output and header
  for (i=0;i<4;i++) {
    h5buf[i]=(char *) malloc(sizeof(char)*nsamp*nsub);
    checkCudaErrors(hipMalloc((void **) &dh5buf[i],sizeof(char)*nsamp*nsub));
  }

  // Allocate output buffers
  checkCudaErrors(hipMalloc((void **) &dfbuf,sizeof(float)*nsamp*nsub));
  cbuf=(unsigned char *) malloc(sizeof(unsigned char)*msamp*mchan/ndec);
  checkCudaErrors(hipMalloc((void **) &dcbuf,sizeof(unsigned char)*msamp*mchan/ndec));

  // Allocate DMs and copy to device
  dm=(float *) malloc(sizeof(float)*ndm);
  for (idm=0;idm<ndm;idm++)
    dm[idm]=dm_start+(float) idm*dm_step;
  checkCudaErrors(hipMalloc((void **) &ddm,sizeof(float)*ndm));
  checkCudaErrors(hipMemcpy(ddm,dm,sizeof(float)*ndm,hipMemcpyHostToDevice));

  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCufftErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub));

  // Generate FFT plan (batch in-place backward FFT)
  idist=mbin;  odist=mbin;  iembed=mbin;  oembed=mbin;  istride=1;  ostride=1;
  checkCufftErrors(hipfftPlanMany(&ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub));

  // Compute chirp
  blocksize.x=32; blocksize.y=32; blocksize.z=1;
  gridsize.x=nsub/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=ndm/blocksize.z+1;
  compute_chirp<<<gridsize,blocksize>>>(h5.fcen,nsub*h5.bwchan,ddm,nchan,nbin,nsub,ndm,dc);

  // Write temporary filterbank header
  file=fopen("/tmp/header.fil","w");
  write_filterbank_header(h5,file);
  fclose(file);
  file=fopen("/tmp/header.fil","r");
  bytes_read=fread(fheader,sizeof(char),1024,file);
  fclose(file);
  
  // Format file names and open
  outfile=(FILE **) malloc(sizeof(FILE *)*ndm);
  for (idm=0;idm<ndm;idm++) {
    sprintf(fname,"%s_cDM%06.2f_P%03d.fil",obsid,dm[idm],part);

    outfile[idm]=fopen(fname,"w");
  }
  
  // Write headers
  for (idm=0;idm<ndm;idm++) {
    // Send header
    fwrite(fheader,sizeof(char),bytes_read,outfile[idm]);
  }

  // Read files
  for (i=0;i<4;i++) {
    rawfile[i]=fopen(h5.rawfname[i],"r");
  }

  // Loop over input file contents
  for (iblock=0;;iblock++) {
    // Read block
    startclock=clock();
    for (i=0;i<4;i++)
      nread=fread(h5buf[i],sizeof(char),nsamp*nsub,rawfile[i])/nsub;
    if (nread==0)
      break;
    printf("Block: %d: Read %lu MB in %.2f s\n",iblock,sizeof(char)*nread*nsub*4/(1<<20),(float) (clock()-startclock)/CLOCKS_PER_SEC);

    // Copy buffers to device
    startclock=clock();
    for (i=0;i<4;i++)
      checkCudaErrors(hipMemcpy(dh5buf[i],h5buf[i],sizeof(char)*nread*nsub,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
    unpack_and_padd<<<gridsize,blocksize>>>(dh5buf[0],dh5buf[1],dh5buf[2],dh5buf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);

    // Perform FFTs
    checkCufftErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1p,(hipfftComplex *) cp1p,HIPFFT_FORWARD));
    checkCufftErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2p,(hipfftComplex *) cp2p,HIPFFT_FORWARD));

    // Swap spectrum halves for large FFTs
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft*nsub/blocksize.y+1; gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1p,cp2p,nbin,nfft*nsub);

    // Loop over dms
    for (idm=0;idm<ndm;idm++) {

      // Perform complex multiplication of FFT'ed data with chirp
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=nbin*nsub/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=1;
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1p,dc,cp1,nbin*nsub,nfft,idm,1.0/(float) nbin);
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2p,dc,cp2,nbin*nsub,nfft,idm,1.0/(float) nbin);
      
      // Swap spectrum halves for small FFTs
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan*nfft*nsub/blocksize.y+1; gridsize.z=1;
      swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan*nfft*nsub);
      
      // Perform FFTs
      checkCufftErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
      checkCufftErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
      
      // Detect data
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=nfft/blocksize.z+1;
      transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan,nfft,nsub,noverlap/nchan,nread/nchan,dfbuf);
      
      // Compute block sums for redigitization
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      compute_block_sums<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,bs1,bs2);
      
      // Compute channel stats
      blocksize.x=32; blocksize.y=1; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=1; gridsize.z=1;
      compute_channel_statistics<<<gridsize,blocksize>>>(mchan,mblock,msum,bs1,bs2,zavg,zstd);

      // Redigitize data to 8bits
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      if (ndec==1)
	redigitize<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);
      else
	decimate_and_redigitize<<<gridsize,blocksize>>>(dfbuf,ndec,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);      

      // Copy buffer to host
      checkCudaErrors(hipMemcpy(cbuf,dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost));

      // Write buffer
      fwrite(cbuf,sizeof(char),nread*nsub/ndec,outfile[idm]);
    }
    printf("Processed %d DMs in %.2f s\n",ndm,(float) (clock()-startclock)/CLOCKS_PER_SEC);
  }

  // Close files
  for (i=0;i<ndm;i++)
    fclose(outfile[i]);

  // Close files
  for (i=0;i<4;i++)
    fclose(rawfile[i]);

  // Free
  for (i=0;i<4;i++) {
    free(h5buf[i]);
    hipFree(dh5buf);
    free(h5.rawfname[i]);
  }
  free(dm);
  free(cbuf);
  free(outfile);

  hipFree(dfbuf);
  hipFree(dcbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(cp1p);
  hipFree(cp2p);
  hipFree(dc);
  hipFree(bs1);
  hipFree(bs2);
  hipFree(zavg);
  hipFree(zstd);
  hipFree(ddm);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}

// Usage
void usage()
{
  printf("cdmt -P <part> -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -o <outputname> <file.h5>\n\n");
  printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in HDF5 format.\n");
  printf("-P <part>        Specify part number for input file [integer, default: 0]\n");
  printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
  printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
  printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
  printf("-o <outputname>           Output filename [default: cdmt]\n");
  printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
  printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");

  return;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i+nx*l]),scale);
  }
}

// Compute chirp
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c)
{
  int ibin,ichan,isub,idm,mbin,idx;
  double s,rt,t,f,fsub,fchan,bwchan,bwsub;

  // Number of channels per subband
  mbin=nbin/nchan;

  // Subband bandwidth
  bwsub=bw/nsub;

  // Channel bandwidth
  bwchan=bw/(nchan*nsub);

  // Indices of input data
  isub=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  idm=blockIdx.z*blockDim.z+threadIdx.z;

  // Keep in range
  if (isub<nsub && ichan<nchan && idm<ndm) {
    // Main constant
    s=2.0*M_PI*dm[idm]/DMCONSTANT;

    // Frequencies
    fsub=fcen-0.5*bw+bw*(float) isub/(float) nsub+0.5*bw/(float) nsub;
    fchan=fsub-0.5*bwsub+bwsub*(float) ichan/(float) nchan+0.5*bwsub/(float) nchan;
      
    // Loop over bins in channel
    for (ibin=0;ibin<mbin;ibin++) {
      // Bin frequency
      f=-0.5*bwchan+bwchan*(float) ibin/(float) mbin+0.5*bwchan/(float) mbin;
      
      // Phase delay
      rt=-f*f*s/((fchan+f)*fchan*fchan);
      
      // Taper
      t=1.0/sqrt(1.0+pow((f/(0.47*bwchan)),80));
      
      // Index
      idx=ibin+ichan*mbin+isub*mbin*nchan+idm*nsub*mbin*nchan;
      
      // Chirp
      c[idx].x=cos(rt)*t;
      c[idx].y=sin(rt)*t;
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    idx1=ibin+nbin*isub+nsub*nbin*ifft;
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    idx2=isub+nsub*isamp;
    if (isamp<0 || isamp>=nsamp) {
      cp1[idx1].x=0.0;
      cp1[idx1].y=0.0;
      cp2[idx1].x=0.0;
      cp2[idx1].y=0.0;
    } else {
      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf)
{
  int64_t ibin,ichan,ifft,isub,isamp,idx1,idx2;
  
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  ifft=blockIdx.z*blockDim.z+threadIdx.z;
  if (ibin<nbin && ichan<nchan && ifft<nfft) {
    // Loop over subbands
    for (isub=0;isub<nsub;isub++) {
      // Padded array index
      //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
      idx1=ibin+ichan*nbin+(nsub-isub-1)*nbin*nchan+ifft*nbin*nchan*nsub;

      // Time index
      isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
      
      // Output array index
      idx2=(nchan-ichan-1)+isub*nchan+nsub*nchan*isamp;
      
      // Select data points from valid region
      if (ibin>=noverlap && ibin<=nbin-noverlap && isamp>=0 && isamp<nsamp)
	fbuf[idx2]=cp1[idx1].x*cp1[idx1].x+cp1[idx1].y*cp1[idx1].y+cp2[idx1].x*cp2[idx1].x+cp2[idx1].y*cp2[idx1].y;
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2)
{
  int64_t ichan,iblock,isum,idx1,idx2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    idx1=ichan+nchan*iblock;
    bs1[idx1]=0.0;
    bs2[idx1]=0.0;
    for (isum=0;isum<nsum;isum++) {
      idx2=ichan+nchan*(isum+iblock*nsum);
      bs1[idx1]+=z[idx2];
      bs2[idx1]+=z[idx2]*z[idx2];
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd)
{
  int64_t ichan,iblock,idx1;
  double s1,s2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  if (ichan<nchan) {
    s1=0.0;
    s2=0.0;
    for (iblock=0;iblock<nblock;iblock++) {
      idx1=ichan+nchan*iblock;
      s1+=bs1[idx1];
      s2+=bs2[idx1];
    }
    zavg[ichan]=s1/(float) (nblock*nsum);
    zstd[ichan]=s2/(float) (nblock*nsum)-zavg[ichan]*zavg[ichan];
    zstd[ichan]=sqrt(zstd[ichan]);
  }

  return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1;
  float zoffset,zscale;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum++) {
      idx1=ichan+nchan*(isum+iblock*nsum);
      z[idx1]-=zoffset;
      z[idx1]*=256.0/zscale;
      cz[idx1]=(unsigned char) z[idx1];
      if (z[idx1]<0.0) cz[idx1]=0;
      if (z[idx1]>255.0) cz[idx1]=255;
    }
  }

  return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float zoffset,zscale,ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
	idx1=ichan+nchan*(isum+idec+iblock*nsum);
	ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      ztmp-=zoffset;
      ztmp*=256.0/zscale;
      cz[idx2]=(unsigned char) ztmp;
      if (ztmp<0.0) cz[idx2]=0;
      if (ztmp>255.0) cz[idx2]=255;
    }
  }

  return;
}
